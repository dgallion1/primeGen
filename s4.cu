#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <primesieve.hpp>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <algorithm>
#include <iomanip>
#include <nvml.h>
#include <unistd.h>
#include <chrono>
#include <thread>

__device__ double atomicAddDouble(double* address, double value) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(__longlong_as_double(assumed) + value));
    } while (assumed != old);
    return __longlong_as_double(old);
}

__global__ void computeSum(double* partialSums, const char* signs, long long start, long long end, long long maxN) {
    extern __shared__ double shared[];  // Shared memory allocation
    long long i = blockIdx.x * blockDim.x + threadIdx.x + start;
    int tid = threadIdx.x;

    // Initialize shared memory with the computed term for each thread
    double term = 0.0;
    if (i <= maxN && i <= end) {
        term = 1.0 / i;
        if (signs[i - start] == 0 && i > 4) {
            term = -term;
        }
    }
    shared[tid] = term;

    // Synchronize to make sure all threads have written to shared memory
    __syncthreads();

    // Reduction in shared memory to compute the block-level partial sum using Kahan summation
    double c = 0.0;  // Compensation for lost low-order bits
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            double y = shared[tid + s] - c;
            double t = shared[tid] + y;
            c = (t - shared[tid]) - y;
            shared[tid] = t;
        }
        // Synchronize to ensure all threads have completed this step of reduction
        __syncthreads();
    }

    // Write the result from the first thread of each block to global memory
    if (tid == 0) {
        partialSums[blockIdx.x] = shared[0];
    }
}

void calculateSumInChunks(long long highestPrime, long long maxN, int chunkSize) {
    double totalResult = 0.0;
    long long start = 0;

    // Set CUDA device
    if (hipSetDevice(0) != hipSuccess) {
        std::cerr << "Error setting CUDA device." << std::endl;
        return;
    }

    // Allocate signs array on GPU (reuse for each chunk)
    char* d_signs;
    if (hipMalloc((void**)&d_signs, chunkSize * sizeof(char)) != hipSuccess) {
        std::cerr << "Error allocating memory for signs on GPU." << std::endl;
        return;
    }

    // Allocate partial sums array on GPU (reuse for each chunk)
    double* d_partialSums;
    int blks = 1024;  // Number of threads per block
    int maxBlocksPerGrid = 65535;
    int blocksPerGrid = std::min((chunkSize + blks - 1) / blks, maxBlocksPerGrid);
    if (hipMalloc((void**)&d_partialSums, blocksPerGrid * sizeof(double)) != hipSuccess) {
        std::cerr << "Error allocating memory for partial sums on GPU." << std::endl;
        hipFree(d_signs);
        return;
    }

    // Create CUDA streams
    const int numStreams = 4;
    hipStream_t streams[numStreams];
    for (int i = 0; i < numStreams; ++i) {
        hipStreamCreate(&streams[i]);
    }

    // Process chunks of the range from 0 to highestPrime
    while (start < highestPrime) {
        long long end = std::min(start + chunkSize, highestPrime);
        std::vector<long long> primes;
        primesieve::generate_primes(start + 1, end, &primes);
        int numPrimes = primes.size();

        // Error checking for empty prime chunk
        if (numPrimes == 0) {
            std::cerr << "No primes found in the current chunk: start = " << start << ", end = " << end << std::endl;
            start = end;
            continue;
        }

        // Create signs array on host
        std::vector<char> signs(chunkSize, 1);
        bool currentSign = true;  // Start with positive sign
        long long previousPrimeIndex = 0;
        for (int i = 0; i < numPrimes; ++i) {
            long long primeIndex = primes[i] - (start + 1);
            for (long long j = previousPrimeIndex; j < primeIndex; ++j) {
                signs[j] = currentSign ? 1 : 0;
            }
            currentSign = !currentSign;  // Toggle the sign after each prime
            previousPrimeIndex = primeIndex;
        }
        // Fill the remaining signs after the last prime in the chunk
        for (long long j = previousPrimeIndex; j < chunkSize; ++j) {
            signs[j] = currentSign ? 1 : 0;
        }

        // Use multiple streams to copy data and launch kernels
        int streamIdx = (start / chunkSize) % numStreams;

        // Copy signs array to device asynchronously
        if (hipMemcpyAsync(d_signs, signs.data(), chunkSize * sizeof(char), hipMemcpyHostToDevice, streams[streamIdx]) != hipSuccess) {
            std::cerr << "Error copying signs to GPU." << std::endl;
            hipFree(d_partialSums);
            hipFree(d_signs);
            for (int i = 0; i < numStreams; ++i) {
                hipStreamDestroy(streams[i]);
            }
            return;
        }

        // Launch kernel asynchronously
        computeSum<<<blocksPerGrid, blks, blks * sizeof(double), streams[streamIdx]>>>(d_partialSums, d_signs, start + 1, end, maxN);

        // Copy partial sums from GPU to host asynchronously
        std::vector<double> partialSums(blocksPerGrid, 0.0);
        if (hipMemcpyAsync(partialSums.data(), d_partialSums, blocksPerGrid * sizeof(double), hipMemcpyDeviceToHost, streams[streamIdx]) != hipSuccess) {
            std::cerr << "Error copying partial sums from GPU." << std::endl;
            hipFree(d_partialSums);
            hipFree(d_signs);
            for (int i = 0; i < numStreams; ++i) {
                hipStreamDestroy(streams[i]);
            }
            return;
        }

        // Wait for all streams to complete before accumulating the result
        for (int i = 0; i < numStreams; ++i) {
            hipStreamSynchronize(streams[i]);
        }

        // Sum partial sums on the host using Kahan summation to reduce numerical error
        double c = 0.0;  // Compensation for lost low-order bits
        for (double partialSum : partialSums) {
            double y = partialSum - c;
            double t = totalResult + y;
            c = (t - totalResult) - y;
            totalResult = t;
        }

        // Update the start for the next chunk
        start = end;
    }

    // Clean up GPU memory
    hipFree(d_partialSums);
    hipFree(d_signs);
    for (int i = 0; i < numStreams; ++i) {
        hipStreamDestroy(streams[i]);
    }

    std::cout << std::fixed << std::setprecision(15) << "Resulting sum: " << totalResult << std::endl;

    // Reset CUDA device
    hipDeviceReset();
}

int main(int argc, char* argv[]) {
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <highest prime to generate>" << std::endl;
        return 1;
    }

    long long highestPrime = std::atoll(argv[1]);
    if (highestPrime <= 0) {
        std::cerr << "Please provide a positive number for the highest prime to generate." << std::endl;
        return 1;
    }

    // Set the maximum value of N for the calculation
    long long maxN = highestPrime;

    // Define chunk size for handling large number of primes (increased to improve GPU utilization)
    int chunkSize = 10000000;  // Further increased chunk size to improve GPU utilization

    // Call calculateSumInChunks to handle large prime arrays in chunks
    calculateSumInChunks(highestPrime, maxN, chunkSize);

    return 0;
}